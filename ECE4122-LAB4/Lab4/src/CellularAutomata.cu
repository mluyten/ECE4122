#include "hip/hip_runtime.h"
/*
Author: Matthew Luyten
Class: ECE4122
Last Date Modified: 11/6/2024

Description:
This implements the LifeManager class and all cellular automata functions for use on the GPU.
*/

#include "CellularAutomata.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

// Checks adjacent cells to cell at lastGeneration[r][c] and sets its value at thisGeneration[r][c]
// This function uses the cuda thread id to determine its position (row, col)
// thisGeneration - pointer to most recent generation of automata - this is set by this function
// lastGeneration - pointer to last generation of automata - this is read by the function and determines nextGeneration
// rows - number of rows in the simulation
// cols - number of cols in the simulation
__global__ void ageCell (int* thisGeneration, const int* lastGeneration, int rows, int cols) {
    // Get cuda thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we're not out of bounds
    if (i < rows * cols) {
        // Determine (row, col) postion
        int r = i / cols;
        int c = i - r * cols;
        int sum = 0;
        // Iterates through all cells adjacent to desired position and sums number of live cells
        for (int i(r > 0 ? -1 : 0); i <= int(r < rows - 1 ? 1 : 0); i++) {
            for (int j(c > 0 ? -1 : 0); j <= int(c < cols - 1 ? 1 : 0); j++) {
                // Ignores desired cell
                if (j != 0 || i != 0)
                    sum += lastGeneration[(r + i) * cols + c + j];
            }
        }

        // Game of life logic
        if (lastGeneration[r * cols + c] && (sum < 2 || sum > 3))
            thisGeneration[r * cols + c] = 0;
        else if (!lastGeneration[r * cols + c] && sum == 3)
            thisGeneration[r * cols + c] = 1;
        else
            thisGeneration[r * cols + c] = lastGeneration[r * cols + c];
    }
    return;
}

// This function progresses the simulation by one iteration. It offloads that computation to the GPU
// nThreads - number of threads per block to be used on the GPU
int LifeManager::nextGeneration(int nThreads) {
    // Start wall timer
    auto tick = std::chrono::high_resolution_clock::now();
    // Get total size of array
    size_t size = _rows * _cols * sizeof(int);

    // Set up thread blocks based on nThreads value
    int threadsPerBlock = nThreads;
    int blocksPerGrid = (_rows * _cols + threadsPerBlock - 1) / threadsPerBlock;

    // Make sure any async coppies are done
    hipDeviceSynchronize();
    
    // Run ageCell on the GPU
    ageCell << <blocksPerGrid, threadsPerBlock >> > (_d_thisGeneration, _d_lastGeneration, _rows, _cols);

    // Make sure all computations are finished
    hipDeviceSynchronize();
    
    if (_mode != 2) {
        // If not using managed memory, async copy next generation for next iteration and copy result to host
        hipMemcpyAsync(_d_lastGeneration, _d_thisGeneration, size, hipMemcpyDeviceToDevice);
        hipMemcpy(_thisGeneration, _d_thisGeneration, size, hipMemcpyDeviceToHost);
    }
    else {
        // Copy result to last generation for next iteration
        hipMemcpy(_d_lastGeneration, _d_thisGeneration, size, hipMemcpyDeviceToDevice);
    }

    // Stop wall timer
    auto tock = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::microseconds>(tock - tick).count();
}

// Constructor for LifeManager class. Allocates memory based on GPU memory mode.
// rows - number of rows in simulation
// cols - number of columns in simulation
// mode - memory mode for the simulation (0 - NORMAL, 1 - PINNED, 2 - MANAGED)
LifeManager::LifeManager(int rows, int cols, int mode) : _rows(rows), _cols(cols), _mode(mode) {
    if (_mode == 0) {
        // Alloc buffers on host
        _thisGeneration = new int[rows * cols];
        _lastGeneration = new int[rows * cols];
        // Alloc buffers on device
        hipMalloc(&_d_thisGeneration, rows * cols * sizeof(int));
        hipMalloc(&_d_lastGeneration, rows * cols * sizeof(int));
    }
    else if (_mode == 1) {
        // Alloc pinned buffers on host
        hipHostMalloc(&_thisGeneration, rows * cols * sizeof(int));
        hipHostMalloc(&_lastGeneration, rows * cols * sizeof(int));
        // Alloc buffers on device
        hipMalloc(&_d_thisGeneration, rows * cols * sizeof(int));
        hipMalloc(&_d_lastGeneration, rows * cols * sizeof(int));
    }
    else {
        // Alloc managed buffers
        hipMallocManaged(&_thisGeneration, rows * cols * sizeof(int));
        hipMallocManaged(&_lastGeneration, rows * cols * sizeof(int));
        // Set host/device pointers to same value since they are the same buffers
        _d_thisGeneration = _thisGeneration;
        _d_lastGeneration = _lastGeneration;
    }
}

// Destructor for LifeManager class. Frees up any buffers made on init.
LifeManager::~LifeManager() {
    if (_mode == 0) {
        free(_thisGeneration);
        free(_lastGeneration);
        hipFree(_d_thisGeneration);
        hipFree(_d_lastGeneration);
    }
    else if (_mode == 1) {
        hipHostFree(_thisGeneration);
        hipHostFree(_lastGeneration);
        hipFree(_d_thisGeneration);
        hipFree(_d_lastGeneration);
    }
    else {
        hipFree(_thisGeneration);
        hipFree(_lastGeneration);
    }
}

// This function gets the value at position (r, c) in the must recent generation of life.
// r - Row position
// c - Column position
int LifeManager::get(int r, int c) {
    return _thisGeneration[r * _cols + c];
}

// Sets the value at position (r, c) in the last generation of life. This is used to initialize the values
// at the start of the simulation.
// r - Row position
// c - Column position
void LifeManager::set(int r, int c, int val) {
    _lastGeneration[r * _cols + c] = val;
}

// Initializes the simulation by copying the lastGeneration buffer onto the GPU (if not using managed memory).
// This is functionally a noop if using MANAGED memory mode.
// r - Row position
// c - Column position
void LifeManager::init() {
    if (_mode != 2)
        hipMemcpy(_d_lastGeneration, _lastGeneration, _rows * _cols * sizeof(int), hipMemcpyHostToDevice);
    return;
}